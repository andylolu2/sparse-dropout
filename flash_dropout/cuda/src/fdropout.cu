#include <cute/tensor.hpp>
//
#include <cutlass/util/device_memory.h>
#include <torch/extension.h>

#include <vector>

#include "mask_utils.cuh"
#include "matmul_dsd.cuh"
#include "matmul_sdd.cuh"

#ifndef JIT_BLK_M_0
#define JIT_BLK_M_0 128
#endif
#ifndef JIT_BLK_N_0
#define JIT_BLK_N_0 128
#endif
#ifndef JIT_BLK_K_0
#define JIT_BLK_K_0 64
#endif
#ifndef JIT_GROUP_0
#define JIT_GROUP_0 6
#endif
#ifndef JIT_BLK_M_1
#define JIT_BLK_M_1 128
#endif
#ifndef JIT_BLK_N_1
#define JIT_BLK_N_1 128
#endif
#ifndef JIT_BLK_K_1
#define JIT_BLK_K_1 64
#endif
#ifndef JIT_GROUP_1
#define JIT_GROUP_1 6
#endif
#ifndef JIT_BLK_M_2
#define JIT_BLK_M_2 128
#endif
#ifndef JIT_BLK_N_2
#define JIT_BLK_N_2 128
#endif
#ifndef JIT_BLK_K_2
#define JIT_BLK_K_2 64
#endif
#ifndef JIT_GROUP_2
#define JIT_GROUP_2 6
#endif

namespace ct = cute;

template <typename T, typename Stride>
auto torch_to_ct_2d(torch::Tensor x) {
    auto ptr = ct::make_gmem_ptr(reinterpret_cast<T *>(x.data_ptr()));
    return ct::make_tensor(ptr, ct::make_shape(x.size(0), x.size(1)), Stride{});
}

void check_half_cuda_2d_row_major(torch::Tensor x) {
    TORCH_CHECK(x.dtype() == torch::kHalf, "X must be half");
    TORCH_CHECK(x.type().is_cuda(), "X must be CUDA");
    TORCH_CHECK(x.ndimension() == 2, "X must be 2D");
    TORCH_CHECK(x.stride(0) == x.size(1) && x.stride(1) == 1, "X must be contiguous and row major");
}

// C++ interface

// --- forward ---
template <typename KernelTraits>
torch::Tensor forward_core(torch::Tensor A, torch::Tensor B, float p, torch::Tensor mask) {
    check_half_cuda_2d_row_major(A);
    check_half_cuda_2d_row_major(B);
    TORCH_CHECK(B.size(1) == A.size(1), "B must have the same K dim size as A");

    int64_t M = A.size(0);
    int64_t N = B.size(0);
    auto C = torch::empty({M, N}, A.options());  // Allocate output tensor

    auto A_ct = torch_to_ct_2d<ct::half_t, ct::GenRowMajor>(A);
    auto B_ct = torch_to_ct_2d<ct::half_t, ct::GenRowMajor>(B);
    auto C_ct = torch_to_ct_2d<ct::half_t, ct::GenRowMajor>(C);
    auto mask_ct = torch_to_ct_2d<ct::uint8_t, ct::GenRowMajor>(mask);

    matmul_dsd<KernelTraits>(A_ct, B_ct, C_ct, mask_ct, static_cast<ct::half_t>(1 / (1 - p)));
    return C;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, int64_t> forward(
    torch::Tensor A, torch::Tensor B, float p) {
    using KernelTraits = KernelTraits<ct::half_t, JIT_BLK_M_0, JIT_BLK_N_0, JIT_BLK_K_0,
                                      JIT_GROUP_0, true, true, true>;

    constexpr int BLK_M = std::lcm(JIT_BLK_M_0, std::lcm(JIT_BLK_M_1, JIT_BLK_M_2));
    constexpr int BLK_K = std::lcm(JIT_BLK_K_0, std::lcm(JIT_BLK_K_1, JIT_BLK_K_2));
    auto [mask, mask_T, mask_table, count] = make_mask(A.size(0), A.size(1), 128, p);
    // make_mask<BLK_M, BLK_K, JIT_BLK_M_0, JIT_BLK_K_0, JIT_BLK_M_1, JIT_BLK_K_1, JIT_BLK_M_2,
    //   JIT_BLK_K_2>(A.size(0), A.size(1), p);
    auto C = forward_core<KernelTraits>(A, B, p, mask);
    return {C, mask, mask_T, mask_table, count};
}

// Debugging interface
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, int64_t> forward_test(
    torch::Tensor A, torch::Tensor B, torch::Tensor m, float p) {
    using KernelTraits = KernelTraits<ct::half_t, JIT_BLK_M_0, JIT_BLK_N_0, JIT_BLK_K_0,
                                      JIT_GROUP_0, true, true, true>;

    constexpr int BLK_M = std::lcm(JIT_BLK_M_0, std::lcm(JIT_BLK_M_1, JIT_BLK_M_2));
    constexpr int BLK_K = std::lcm(JIT_BLK_K_0, std::lcm(JIT_BLK_K_1, JIT_BLK_K_2));
    auto [mask, mask_T, mask_table, count] =
        make_mask_from_existing<BLK_M, BLK_K, JIT_BLK_M_0, JIT_BLK_K_0, JIT_BLK_M_1, JIT_BLK_K_1,
                                JIT_BLK_M_2, JIT_BLK_K_2>(m);
    auto C = forward_core<KernelTraits>(A, B, p, mask);
    return {C, mask, mask_T, mask_table, count};
}

torch::Tensor backward_dA(torch::Tensor dC, torch::Tensor B, torch::Tensor mask_table, float p,
                          int64_t count) {
    check_half_cuda_2d_row_major(dC);
    check_half_cuda_2d_row_major(B);
    TORCH_CHECK(dC.size(1) == B.size(0), "dC must have the same N dim size as B");
    TORCH_CHECK(mask_table.size(1) == 2, "mask_table should an array of 2D coordinates")

    int64_t M = dC.size(0);
    int64_t K = B.size(1);
    auto dA = torch::zeros({M, K}, B.options());

    auto B_T_ct = torch_to_ct_2d<ct::half_t, ct::GenColMajor>(B.t());
    auto dA_ct = torch_to_ct_2d<ct::half_t, ct::GenRowMajor>(dA);
    auto dC_ct = torch_to_ct_2d<ct::half_t, ct::GenRowMajor>(dC);

    using KernelTraits = KernelTraits<ct::half_t, JIT_BLK_M_1, JIT_BLK_K_1, JIT_BLK_N_1,
                                      JIT_GROUP_1, true, false, true>;
    auto mask_ptr = ct::make_gmem_ptr(reinterpret_cast<ct::int64_t *>(mask_table.data_ptr()));
    auto mask_table_ct =
        ct::make_tensor(mask_ptr, ct::make_shape(count, Int<2>{}), ct::GenRowMajor{});
    matmul_sdd<KernelTraits>(dC_ct, B_T_ct, dA_ct, mask_table_ct, count,
                             static_cast<ct::half_t>(1 / (1 - p)));

    return dA;
}

torch::Tensor backward_dB(torch::Tensor dC, torch::Tensor A, torch::Tensor mask_T, float p) {
    check_half_cuda_2d_row_major(dC);
    check_half_cuda_2d_row_major(A);
    TORCH_CHECK(dC.size(0) == A.size(0), "dC must have the same M dim size as A");

    int64_t N = dC.size(1);
    int64_t K = A.size(1);
    auto dB = torch::empty({N, K}, A.options());

    auto A_T_ct = torch_to_ct_2d<ct::half_t, ct::GenColMajor>(A.t());
    auto dB_T_ct = torch_to_ct_2d<ct::half_t, ct::GenColMajor>(dB.t());
    auto dC_T_ct = torch_to_ct_2d<ct::half_t, ct::GenColMajor>(dC.t());
    auto mask_T_ct = torch_to_ct_2d<ct::uint8_t, ct::GenRowMajor>(mask_T);

    using KernelTraits = KernelTraits<ct::half_t, JIT_BLK_K_2, JIT_BLK_N_2, JIT_BLK_M_2,
                                      JIT_GROUP_2, false, false, false>;
    matmul_dsd<KernelTraits>(A_T_ct, dC_T_ct, dB_T_ct, mask_T_ct,
                             static_cast<ct::half_t>(1 / (1 - p)));

    return dB;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward);
    m.def("forward_test", &forward_test);
    m.def("backward_dA", &backward_dA);
    m.def("backward_dB", &backward_dB);
}